#include "iwp.hxx"

int iwp::get1DCoords(cv::Mat &img, pixel_coords coords)
{
    return (coords.second * img.size().width) + coords.first;
}

pixel_coords iwp::get2DCoords(int width, int coord)
{
    return pixel_coords(coord % width, coord / width);
}

std::vector<int> iwp::getPixelNeighbours(cv::Mat &img, pixel_coords coords)
{
    std::vector<int> neighbours;

    int floorX = coords.first;
    int floorY = coords.second;
    if (coords.first != 0)
    {
        floorX -= 1;
    }
    if (coords.second != 0)
    {
        floorY -= 1;
    }

    for (int i = floorX; i < coords.first + 2; i++)
    {
        for (int j = floorY; j < coords.second + 2; j++)
        {
            if (!(i == coords.first && j == coords.second) && (i < img.size().width) && (j < img.size().height))
            {
                neighbours.push_back(get1DCoords(img, pixel_coords(i, j)));
            }
        }
    }

    return neighbours;
}

template <typename vertex_t, typename edge_t, typename weight_t>
auto iwp::convertImgToGraph(cv::Mat &marker, cv::Mat &mask, vertex_t *markerValues, vertex_t *maskValues)
{
    using csr_t = gunrock::format::csr_t<gunrock::memory_space_t::device, vertex_t, edge_t, weight_t>;

    const int HAS_EDGE = 1;

    if (marker.empty())
        throw "Unable to read image";

    std::vector<int> columnIdx, rowOffset, values;
    rowOffset.push_back(0);

    for (int i = 0; i < marker.rows; i++)
    {
        for (int j = 0; j < marker.cols; j++)
        {
            pixel_coords pixel = pixel_coords(i, j);
            int oneDPos = get1DCoords(marker, pixel);

            markerValues[oneDPos] = marker.at<uchar>(i, j);
            maskValues[oneDPos] = mask.at<uchar>(i, j);

            std::vector<int> neighbours = getPixelNeighbours(marker, pixel);
            for (int neighbour : neighbours)
            {
                columnIdx.push_back(neighbour);
                values.push_back(HAS_EDGE);
            }
            rowOffset.push_back(rowOffset.back() + neighbours.size());
        }
    }

    csr_t csr(marker.rows * marker.cols, marker.rows * marker.cols, values.size());
    csr.row_offsets = rowOffset;
    csr.column_indices = columnIdx;
    csr.nonzero_values = values;

    // Build graph
    auto G = gunrock::graph::build::from_csr<gunrock::memory::memory_space_t::device, gunrock::graph::view_t::csr>(
        csr.number_of_rows,              // rows
        csr.number_of_columns,           // columns
        csr.number_of_nonzeros,          // nonzeros
        csr.row_offsets.data().get(),    // row_offsets
        csr.column_indices.data().get(), // column_indices
        csr.nonzero_values.data().get()  // values
    );

    return G;
}

float iwp::runMorphRec(cv::Mat &marker, cv::Mat &mask)
{
    using vertex_t = int;
    using edge_t = int;
    using weight_t = float;

    int numVertices = marker.rows * marker.cols;

    thrust::device_vector<vertex_t> markerValues(numVertices);
    thrust::device_vector<vertex_t> maskValues(numVertices);

    auto markerGraph = convertImgToGraph<vertex_t, edge_t, weight_t>(marker,
                                                                     mask,
                                                                     markerValues.data().get(),
                                                                     maskValues.data().get());

    float gpu_elapsed = run(markerGraph, maskValues.data().get(), markerValues.data().get());

    std::cout << "GPU Elapsed: " << gpu_elapsed << std::endl;
}